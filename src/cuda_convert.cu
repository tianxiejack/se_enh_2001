#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */
#include <math.h>
#include <opencv2/core/core.hpp>
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include "osa_mutex.h"
#include "cuda_convert.cuh"
//#include "enh.hpp"
//#include <ImagesCPU.h>
//#include <ImagesNPP.h>
//#include <ImageIO.h>
//#include <Exceptions.h>
#include <npp.h>
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace cv;

#define GETYVAL(yuv)	(yuv & 0xff)
#define GETUVAL(yuv)	(((yuv)>>8) & 0xff)
#define GETVVAL(yuv)	(((yuv)>>16) & 0xff)

__device__ inline float clamp(float val, float mn, float mx)
{
	return (val >= mn)? ((val <= mx)? val : mx) : mn;
}

__global__ void gpuConvertYUYVtoRGB_kernel(unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	for (int i = 0; i < height; ++i) {
		int y0 = src[i*width*2+idx*4+0];
		int cb = src[i*width*2+idx*4+1];
		int y1 = src[i*width*2+idx*4+2];
		int cr = src[i*width*2+idx*4+3];

		dst[i*width*3+idx*6+0] = clamp(1.164f * (y0 - 16) + 1.596f * (cr - 128)                      , 0.0f, 255.0f);
		dst[i*width*3+idx*6+1] = clamp(1.164f * (y0 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128), 0.0f, 255.0f);
		dst[i*width*3+idx*6+2] = clamp(1.164f * (y0 - 16)                       + 2.018f * (cb - 128), 0.0f, 255.0f);

		dst[i*width*3+idx*6+3] = clamp(1.164f * (y1 - 16) + 1.596f * (cr - 128)                      , 0.0f, 255.0f);
		dst[i*width*3+idx*6+4] = clamp(1.164f * (y1 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128), 0.0f, 255.0f);
		dst[i*width*3+idx*6+5] = clamp(1.164f * (y1 - 16)                       + 2.018f * (cb - 128), 0.0f, 255.0f);
	}
}

void gpuConvertYUYVtoRGB(unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height)
{
	unsigned char *d_src = NULL;
	unsigned char *d_dst = NULL;
	size_t planeSize = width * height * sizeof(unsigned char);

	unsigned int flags;
	bool srcIsMapped = (hipHostGetFlags(&flags, src) == hipSuccess) && (flags & hipHostMallocMapped);
	bool dstIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) && (flags & hipHostMallocMapped);

	if (srcIsMapped) {
		d_src = src;
		hipStreamAttachMemAsync(NULL, src, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_src, planeSize * 2);
		hipMemcpy(d_src, src, planeSize * 2, hipMemcpyHostToDevice);
	}
	if (dstIsMapped) {
		d_dst = dst;
		hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_dst, planeSize * 3);
	}

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	gpuConvertYUYVtoRGB_kernel<<<numBlocks, blockSize>>>(d_src, d_dst, width, height);
	hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachHost);
	hipStreamSynchronize(NULL);

	if (!srcIsMapped) {
		hipMemcpy(dst, d_dst, planeSize * 3, hipMemcpyDeviceToHost);
		hipFree(d_src);
	}
	if (!dstIsMapped) {
		hipFree(d_dst);
	}
}

#if 1
#define DESCALE(x, n)    (((x) + (1 << ((n)-1)))>>(n))
#define COEFFS_0 		(22987)
#define COEFFS_1 		(-11698)
#define COEFFS_2 		(-5636)
#define COEFFS_3 		(29049)
#define clip(minv, maxv, value)  ((value)<minv) ? minv : (((value)>maxv) ? maxv : (value))

__global__ void gpuConvertYUYVtoBGR_kernel(unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	for (int i = 0; i < height; ++i) {
		int y0 = src[i*width*2+idx*4+0];
		int cb = src[i*width*2+idx*4+1];
		int y1 = src[i*width*2+idx*4+2];
		int cr = src[i*width*2+idx*4+3];
		int b = DESCALE((cb - 128)*COEFFS_3, 14);
		int g = DESCALE((cb - 128)*COEFFS_2 + (cr - 128)*COEFFS_1, 14);
		int r = DESCALE((cr - 128)*COEFFS_0, 14);

		dst[i*width*3+idx*6+0] = clip(0, 255, y0 + b);//B
		dst[i*width*3+idx*6+1] = clip(0, 255, y0 + g);//G
		dst[i*width*3+idx*6+2] = clip(0, 255, y0 + r);//R

		dst[i*width*3+idx*6+3] = clip(0, 255, y1 + b);//B
		dst[i*width*3+idx*6+4] = clip(0, 255, y1 + g);//G
		dst[i*width*3+idx*6+5] = clip(0, 255, y1 + r);//R
	}
}
#else
__global__ void gpuConvertYUYVtoBGR_kernel(unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	for (int i = 0; i < height; ++i) {
		int y0 = src[i*width*2+idx*4+0];
		int cb = src[i*width*2+idx*4+1];
		int y1 = src[i*width*2+idx*4+2];
		int cr = src[i*width*2+idx*4+3];

		dst[i*width*3+idx*6+0] = clamp(1.164f * (y0 - 16)                       + 2.018f * (cb - 128), 0.0f, 255.0f);
		dst[i*width*3+idx*6+1] = clamp(1.164f * (y0 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128), 0.0f, 255.0f);
		dst[i*width*3+idx*6+2] = clamp(1.164f * (y0 - 16) + 1.596f * (cr - 128)                      , 0.0f, 255.0f);

		dst[i*width*3+idx*6+3] = clamp(1.164f * (y1 - 16)                       + 2.018f * (cb - 128), 0.0f, 255.0f);
		dst[i*width*3+idx*6+4] = clamp(1.164f * (y1 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128), 0.0f, 255.0f);
		dst[i*width*3+idx*6+5] = clamp(1.164f * (y1 - 16) + 1.596f * (cr - 128)                      , 0.0f, 255.0f);
	}
}
#endif

void gpuConvertYUYVtoBGR(unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height)
{
	unsigned char *d_src = NULL;
	unsigned char *d_dst = NULL;
	size_t planeSize = width * height * sizeof(unsigned char);

	unsigned int flags;
	bool srcIsMapped = (hipHostGetFlags(&flags, src) == hipSuccess) && (flags & hipHostMallocMapped);
	bool dstIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) && (flags & hipHostMallocMapped);

	if (srcIsMapped) {
		d_src = src;
		hipStreamAttachMemAsync(NULL, src, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_src, planeSize * 2);
		hipMemcpy(d_src, src, planeSize * 2, hipMemcpyHostToDevice);
	}
	if (dstIsMapped) {
		d_dst = dst;
		hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_dst, planeSize * 3);
	}

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	gpuConvertYUYVtoBGR_kernel<<<numBlocks, blockSize>>>(d_src, d_dst, width, height);
	hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachHost);
	hipStreamSynchronize(NULL);

	if (!srcIsMapped) {
		hipFree(d_src);
	}

	if (!dstIsMapped) {
		hipMemcpy(dst, d_dst, planeSize * 3, hipMemcpyDeviceToHost);
		hipFree(d_dst);
	}
}

void gpuConvertYUYVtoBGR_dev(unsigned char *src, unsigned char *d_dst,
		unsigned int width, unsigned int height)
{
	unsigned char *d_src = NULL;
	size_t planeSize = width * height * sizeof(unsigned char);

	unsigned int flags;
	bool srcIsMapped = (hipHostGetFlags(&flags, src) == hipSuccess) && (flags & hipHostMallocMapped);

	if (srcIsMapped) {
		d_src = src;
		hipStreamAttachMemAsync(NULL, src, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_src, planeSize * 2);
		hipMemcpy(d_src, src, planeSize * 2, hipMemcpyHostToDevice);
	}

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	gpuConvertYUYVtoBGR_kernel<<<numBlocks, blockSize>>>(d_src, d_dst, width, height);
	hipStreamSynchronize(NULL);

	if (!srcIsMapped) {
		hipFree(d_src);
	}
}

//////////////////////////////////////////////////////////////////////
//
//
__global__ void gpuConvertYUYVtoGray_kernel(unsigned char *src,
		unsigned char *dY,
		unsigned int width, unsigned int height)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	int halfH = height>>1;
	for (int i = 0; i < halfH; ++i) {
		int y00 = src[i*2*width*2+idx*4+0];
		int y01 = src[i*2*width*2+idx*4+2];
		int y10 = src[(i*2+1)*width*2+idx*4+0];
		int y11 = src[(i*2+1)*width*2+idx*4+2];

		dY[i*2*width+idx*2+0] = y00;
		dY[i*2*width+idx*2+1] = y01;
		dY[(i*2+1)*width+idx*2+0] = y10;
		dY[(i*2+1)*width+idx*2+1] = y11;
	}
}

void gpuConvertYUYVtoGray(unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height)
{
	unsigned char *d_src = NULL;
	unsigned char *d_dst = NULL;
	size_t planeSize = width * height * sizeof(unsigned char);

	unsigned int flags;
	bool srcIsMapped = (hipHostGetFlags(&flags, src) == hipSuccess) && (flags & hipHostMallocMapped);
	bool dstIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) && (flags & hipHostMallocMapped);

	if (srcIsMapped) {
		d_src = src;
		hipStreamAttachMemAsync(NULL, src, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_src, planeSize * 2);
		hipMemcpy(d_src, src, planeSize * 2, hipMemcpyHostToDevice);
	}
	if (dstIsMapped) {
		d_dst = dst;
		hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_dst, planeSize);
	}

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	gpuConvertYUYVtoGray_kernel<<<numBlocks, blockSize>>>(d_src,d_dst, width, height);
	hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachHost);
	hipStreamSynchronize(NULL);

	if (!srcIsMapped) {
		hipFree(d_src);
	}
	if (!dstIsMapped) {
		hipMemcpy(dst, d_dst, planeSize, hipMemcpyDeviceToHost);
		hipFree(d_dst);
	}
}


//////////////////////////////////////////////////////////////////////
//
//
__global__ void gpuConvertYUYVtoI420_kernel(unsigned char *src,
		unsigned char *dY,unsigned char *dU,unsigned char *dV,
		unsigned int width, unsigned int height)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	int halfH = height>>1;
	for (int i = 0; i < halfH; ++i) {
		int y00 = src[i*2*width*2+idx*4+0];
		int cb0 = src[i*2*width*2+idx*4+1];
		int y01 = src[i*2*width*2+idx*4+2];
		int cr0 = src[i*2*width*2+idx*4+3];
		int y10 = src[(i*2+1)*width*2+idx*4+0];
		int y11 = src[(i*2+1)*width*2+idx*4+2];

		dY[i*2*width+idx*2+0] = y00;
		dY[i*2*width+idx*2+1] = y01;
		dY[(i*2+1)*width+idx*2+0] = y10;
		dY[(i*2+1)*width+idx*2+1] = y11;
		dU[i*(width>>1)+idx] = cb0;
		dV[i*(width>>1)+idx] = cr0;
	}
}

void gpuConvertYUYVtoI420(unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height)
{
	unsigned char *d_src = NULL;
	unsigned char *d_dst = NULL;
	size_t planeSize = width * height * sizeof(unsigned char);

	unsigned int flags;
	bool srcIsMapped = (hipHostGetFlags(&flags, src) == hipSuccess) && (flags & hipHostMallocMapped);
	bool dstIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) && (flags & hipHostMallocMapped);

	if (srcIsMapped) {
		d_src = src;
		hipStreamAttachMemAsync(NULL, src, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_src, planeSize * 2);
		hipMemcpy(d_src, src, planeSize * 2, hipMemcpyHostToDevice);
	}
	if (dstIsMapped) {
		d_dst = dst;
		hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_dst, planeSize * 3/2);
	}

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	gpuConvertYUYVtoI420_kernel<<<numBlocks, blockSize>>>(d_src,
			d_dst, d_dst+planeSize, d_dst+planeSize+(planeSize>>2), width, height);
	hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachHost);
	hipStreamSynchronize(NULL);

	if (!srcIsMapped) {
		hipFree(d_src);
	}
	if (!dstIsMapped) {
		hipMemcpy(dst, d_dst, planeSize * 3/2, hipMemcpyDeviceToHost);
		hipFree(d_dst);
	}
}


//////////////////////////////////////////////////////////////////////
//
//
__global__ void gpuConvertYUYVtoI420AndOsd_kernel(unsigned char *osd,unsigned char *src,
		unsigned char *dY,unsigned char *dU,unsigned char *dV,
		unsigned int width, unsigned int height,
		int yColor, int uColor, int vColor)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	int halfH = height>>1;
	for (int i = 0; i < halfH; ++i) {
		int y00 = src[i*2*width*2+idx*4+0];
		int cb0 = src[i*2*width*2+idx*4+1];
		int y01 = src[i*2*width*2+idx*4+2];
		int cr0 = src[i*2*width*2+idx*4+3];
		int y10 = src[(i*2+1)*width*2+idx*4+0];
		int y11 = src[(i*2+1)*width*2+idx*4+2];

		int a00 = (osd[i*2*width+idx*2+0]+1)>>8;
		int a01 = (osd[i*2*width+idx*2+1]+1)>>8;
		int a10 = (osd[(i*2+1)*width+idx*2+0]+1)>>8;
		int a11 = (osd[(i*2+1)*width+idx*2+1]+1)>>8;
		int auv = (a00 | a01 | a10 | a11);

		dY[i*2*width+idx*2+0] = (1-a00)*y00 + a00*yColor;
		dY[i*2*width+idx*2+1] = (1-a01)*y01 + a01*yColor;
		dY[(i*2+1)*width+idx*2+0] = (1-a10)*y10 + a10*yColor;
		dY[(i*2+1)*width+idx*2+1] = (1-a11)*y11 + a11*yColor;
		dU[i*(width>>1)+idx] = (1-auv)*cb0 + auv*uColor;
		dV[i*(width>>1)+idx] = (1-auv)*cr0 + auv*vColor;
	}
}

void gpuConvertYUYVtoI420AndOsd(unsigned char  *osd,unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height, int colorYUV)
{
	unsigned char *d_src = NULL;
	unsigned char *d_dst = NULL;
	unsigned char *d_osd = NULL;
	size_t planeSize = width * height * sizeof(unsigned char);

	unsigned int flags;
	bool srcIsMapped = (hipHostGetFlags(&flags, src) == hipSuccess) && (flags & hipHostMallocMapped);
	bool dstIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) && (flags & hipHostMallocMapped);
	bool osdIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) && (flags & hipHostMallocMapped);

	if (srcIsMapped) {
		d_src = src;
		hipStreamAttachMemAsync(NULL, src, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_src, planeSize * 2);
		hipMemcpy(d_src, src, planeSize * 2, hipMemcpyHostToDevice);
	}
	if (osdIsMapped) {
		d_osd = osd;
		hipStreamAttachMemAsync(NULL, osd, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_osd, planeSize * 1);
		hipMemcpy(d_osd, osd, planeSize * 1, hipMemcpyHostToDevice);
	}
	if (dstIsMapped) {
		d_dst = dst;
		hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_dst, planeSize * 3/2);
	}

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	int yColor = GETYVAL(colorYUV);
	int uColor = GETUVAL(colorYUV);
	int vColor = GETVVAL(colorYUV);
	gpuConvertYUYVtoI420AndOsd_kernel<<<numBlocks, blockSize>>>(d_osd,d_src,
			d_dst, d_dst+planeSize, d_dst+planeSize+(planeSize>>2), width, height,
			yColor, uColor, vColor);
	hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachHost);
	hipStreamSynchronize(NULL);

	if (!srcIsMapped) {
		hipFree(d_src);
	}
	if (!osdIsMapped) {
		hipFree(d_osd);
	}
	if (!dstIsMapped) {
		hipMemcpy(dst, d_dst, planeSize * 3/2, hipMemcpyDeviceToHost);
		hipFree(d_dst);
	}
}


//////////////////////////////////
//
//
__global__ void gpuConvertGRAYtoI420_kernel(unsigned char *src,
		unsigned char *dY,unsigned char *dU,unsigned char *dV,
		unsigned int width, unsigned int height)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	int halfH = height>>1;
	for (int i = 0; i < halfH; ++i) {
		int y00 = src[i*2*width+idx*2+0];
		int y01 = src[i*2*width+idx*2+1];
		int y10 = src[(i*2+1)*width+idx*2+0];
		int y11 = src[(i*2+1)*width+idx*2+1];

		dY[i*2*width+idx*2+0] = y00;
		dY[i*2*width+idx*2+1] = y01;
		dY[(i*2+1)*width+idx*2+0] = y10;
		dY[(i*2+1)*width+idx*2+1] = y11;
		dU[i*(width>>1)+idx] = 0x80;
		dV[i*(width>>1)+idx] = 0x80;
	}
}
__global__ void gpuConvertGRAYtoI420AndOsd_kernel(unsigned char *osd,unsigned char *src,
		unsigned char *dY,unsigned char *dU,unsigned char *dV,
		unsigned int width, unsigned int height,
		int yColor, int uColor, int vColor)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	int halfH = height>>1;
	for (int i = 0; i < halfH; ++i) {
		int y00 = src[i*2*width+idx*2+0];
		int y01 = src[i*2*width+idx*2+1];
		int y10 = src[(i*2+1)*width+idx*2+0];
		int y11 = src[(i*2+1)*width+idx*2+1];

		int a00 = (osd[i*2*width+idx*2+0]+1)>>8;
		int a01 = (osd[i*2*width+idx*2+1]+1)>>8;
		int a10 = (osd[(i*2+1)*width+idx*2+0]+1)>>8;
		int a11 = (osd[(i*2+1)*width+idx*2+1]+1)>>8;
		int auv = (a00 | a01 | a10 | a11);

		dY[i*2*width+idx*2+0] = (1-a00)*y00 + a00*yColor;
		dY[i*2*width+idx*2+1] = (1-a01)*y01 + a01*yColor;
		dY[(i*2+1)*width+idx*2+0] = (1-a10)*y10 + a10*yColor;
		dY[(i*2+1)*width+idx*2+1] = (1-a11)*y11 + a11*yColor;
		dU[i*(width>>1)+idx] = (1-auv)*0x80 + auv*uColor;
		dV[i*(width>>1)+idx] = (1-auv)*0x80 + auv*vColor;
	}
}

void gpuConvertGRAYtoI420AndOsd(unsigned char  *osd,unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height, int colorYUV)
{
	unsigned char *d_src = NULL;
	unsigned char *d_dst = NULL;
	unsigned char *d_osd = NULL;
	size_t planeSize = width * height * sizeof(unsigned char);

	unsigned int flags;
	bool srcIsMapped = (hipHostGetFlags(&flags, src) == hipSuccess) && (flags & hipHostMallocMapped);
	bool dstIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) && (flags & hipHostMallocMapped);
	bool osdIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) && (flags & hipHostMallocMapped);

	if (srcIsMapped) {
		d_src = src;
		hipStreamAttachMemAsync(NULL, src, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_src, planeSize * 1);
		hipMemcpy(d_src, src, planeSize * 1, hipMemcpyHostToDevice);
	}
	if (osdIsMapped) {
		d_osd = osd;
		hipStreamAttachMemAsync(NULL, osd, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_osd, planeSize * 1);
		hipMemcpy(d_osd, osd, planeSize * 1, hipMemcpyHostToDevice);
	}
	if (dstIsMapped) {
		d_dst = dst;
		hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_dst, planeSize * 3/2);
	}

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	int yColor = GETYVAL(colorYUV);
	int uColor = GETUVAL(colorYUV);
	int vColor = GETVVAL(colorYUV);
	gpuConvertGRAYtoI420AndOsd_kernel<<<numBlocks, blockSize>>>(d_osd,d_src,
			d_dst, d_dst+planeSize, d_dst+planeSize+(planeSize>>2), width, height,
			yColor, uColor, vColor);
	hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachHost);
	hipStreamSynchronize(NULL);

	if (!srcIsMapped) {
		hipFree(d_src);
	}
	if (!osdIsMapped) {
		hipFree(d_osd);
	}
	if (!dstIsMapped) {
		hipMemcpy(dst, d_dst, planeSize * 3/2, hipMemcpyDeviceToHost);
		hipFree(d_dst);
	}
}

//////////////////////////////////
//         GRAYtoI420AndOsdAndZoomx2
//
__device__ inline int meanValuePix2_gray(unsigned char *img)
{
	return ((img[0]+img[1])>>1);
}
__device__ inline int meanValuePix2_gray(unsigned char *img, int width)
{
	return ((img[0]+img[width])>>1);
}
__device__ inline int meanValuePix4_gray(unsigned char *img, int width)
{
	return ((img[0]+img[1]+img[width]+img[width+1])>>2);
}
__global__ void gpuConvertGRAYtoI420AndZoomx_kernel(unsigned char *src,
		unsigned char *dY,unsigned char *dU,unsigned char *dV,
		unsigned int width, unsigned int height, int zoomxStep)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	int halfH = height>>1;
	int halfIx = idx>>zoomxStep;
	for (int i = 0; i < halfH; ++i) {
		int halfIy = i>>zoomxStep;
		int y00 = src[halfIy*2*width+halfIx*2+0];
		int y01 = meanValuePix2_gray(src+halfIy*2*width+halfIx*2+0);
		int y10 = meanValuePix2_gray(src+(halfIy*2+1)*width+halfIx*2+0, width);
		int y11 = meanValuePix4_gray(src+(halfIy*2+1)*width+halfIx*2+0, width);

		dY[i*2*width+idx*2+0] = y00;
		dY[i*2*width+idx*2+1] = y01;
		dY[(i*2+1)*width+idx*2+0] = y10;
		dY[(i*2+1)*width+idx*2+1] = y11;
		dU[i*(width>>1)+idx] = 0x80;
		dV[i*(width>>1)+idx] = 0x80;
	}
}
__global__ void gpuConvertGRAYtoI420AndZoomxAndOsd_kernel(unsigned char *osd,unsigned char *src,
		unsigned char *dY,unsigned char *dU,unsigned char *dV,
		unsigned int width, unsigned int height, int zoomxStep,
		int yColor, int uColor, int vColor)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	int halfH = height>>1;
	int halfIx = idx>>zoomxStep;
	for (int i = 0; i < halfH; ++i) {
		int halfIy = i>>zoomxStep;
		int y00 = src[halfIy*2*width+halfIx*2+0];
		int y01 = meanValuePix2_gray(src+halfIy*2*width+halfIx*2+0);
		int y10 = meanValuePix2_gray(src+(halfIy*2+1)*width+halfIx*2+0, width);
		int y11 = meanValuePix4_gray(src+(halfIy*2+1)*width+halfIx*2+0, width);

		int a00 = (osd[i*2*width+idx*2+0]+1)>>8;
		int a01 = (osd[i*2*width+idx*2+1]+1)>>8;
		int a10 = (osd[(i*2+1)*width+idx*2+0]+1)>>8;
		int a11 = (osd[(i*2+1)*width+idx*2+1]+1)>>8;
		int auv = (a00 | a01 | a10 | a11);

		dY[i*2*width+idx*2+0] = (1-a00)*y00 + a00*yColor;
		dY[i*2*width+idx*2+1] = (1-a01)*y01 + a01*yColor;
		dY[(i*2+1)*width+idx*2+0] = (1-a10)*y10 + a10*yColor;
		dY[(i*2+1)*width+idx*2+1] = (1-a11)*y11 + a11*yColor;
		dU[i*(width>>1)+idx] = (1-auv)*0x80 + auv*uColor;
		dV[i*(width>>1)+idx] = (1-auv)*0x80 + auv*vColor;
	}
}
__global__ void gpuConvertGRAYtoI420AndZoomx_async_kernel(unsigned char *src,
		unsigned char *dY,unsigned char *dU,unsigned char *dV,
		unsigned int width, unsigned int height, unsigned int heightBegin, int zoomxStep)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	int halfIx = idx>>zoomxStep;
	int halfH = height>>1;
	int halfBegin = heightBegin>>1;
	for (int i = 0; i < halfH; ++i) {
		int halfIy = (i+halfBegin)>>zoomxStep;
		int y00 = src[halfIy*2*width+halfIx*2+0];
		int y01 = meanValuePix2_gray(src+halfIy*2*width+halfIx*2+0);
		int y10 = meanValuePix2_gray(src+(halfIy*2+1)*width+halfIx*2+0, width);
		int y11 = meanValuePix4_gray(src+(halfIy*2+1)*width+halfIx*2+0, width);

		dY[i*2*width+idx*2+0] = y00;
		dY[i*2*width+idx*2+1] = y01;
		dY[(i*2+1)*width+idx*2+0] = y10;
		dY[(i*2+1)*width+idx*2+1] = y11;
		dU[i*(width>>1)+idx] = 0x80;
		dV[i*(width>>1)+idx] = 0x80;
	}
}
__global__ void gpuConvertGRAYtoI420AndZoomxAndOsd_async_kernel(unsigned char *osd,unsigned char *src,
		unsigned char *dY,unsigned char *dU,unsigned char *dV,
		unsigned int width, unsigned int height, unsigned int heightBegin, int zoomxStep,
		int yColor, int uColor, int vColor)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	int halfIx = idx>>zoomxStep;
	int halfH = height>>1;
	int halfBegin = heightBegin>>1;
	for (int i = 0; i < halfH; ++i) {
		int halfIy = (i+halfBegin)>>zoomxStep;
		int y00 = src[halfIy*2*width+halfIx*2+0];
		int y01 = meanValuePix2_gray(src+halfIy*2*width+halfIx*2+0);
		int y10 = meanValuePix2_gray(src+(halfIy*2+1)*width+halfIx*2+0, width);
		int y11 = meanValuePix4_gray(src+(halfIy*2+1)*width+halfIx*2+0, width);

		int a00 = (osd[i*2*width+idx*2+0]+1)>>8;
		int a01 = (osd[i*2*width+idx*2+1]+1)>>8;
		int a10 = (osd[(i*2+1)*width+idx*2+0]+1)>>8;
		int a11 = (osd[(i*2+1)*width+idx*2+1]+1)>>8;
		int auv = (a00 | a01 | a10 | a11);

		dY[i*2*width+idx*2+0] = (1-a00)*y00 + a00*yColor;
		dY[i*2*width+idx*2+1] = (1-a01)*y01 + a01*yColor;
		dY[(i*2+1)*width+idx*2+0] = (1-a10)*y10 + a10*yColor;
		dY[(i*2+1)*width+idx*2+1] = (1-a11)*y11 + a11*yColor;
		dU[i*(width>>1)+idx] = (1-auv)*0x80 + auv*uColor;
		dV[i*(width>>1)+idx] = (1-auv)*0x80 + auv*vColor;
	}
}

void gpuConvertGRAYtoI420AndZoomxAndOsd(unsigned char  *osd,unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height, int zoomx, int colorYUV)
{
	unsigned char *d_src = NULL;
	unsigned char *d_dst = NULL;
	unsigned char *d_osd = NULL;
	size_t planeSize = width * height * sizeof(unsigned char);

	unsigned int flags;
	bool srcIsMapped = (hipHostGetFlags(&flags, src) == hipSuccess) && (flags & hipHostMallocMapped);
	bool dstIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) && (flags & hipHostMallocMapped);
	bool osdIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) && (flags & hipHostMallocMapped);

	if (srcIsMapped) {
		d_src = src;
		hipStreamAttachMemAsync(NULL, src, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_src, planeSize * 1);
		hipMemcpy(d_src, src, planeSize * 1, hipMemcpyHostToDevice);
	}
	if (osdIsMapped) {
		d_osd = osd;
		hipStreamAttachMemAsync(NULL, osd, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_osd, planeSize * 1);
		hipMemcpy(d_osd, osd, planeSize * 1, hipMemcpyHostToDevice);
	}
	if (dstIsMapped) {
		d_dst = dst;
		hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_dst, planeSize * 3/2);
	}

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	int yColor = GETYVAL(colorYUV);
	int uColor = GETUVAL(colorYUV);
	int vColor = GETVVAL(colorYUV);
	gpuConvertGRAYtoI420AndZoomxAndOsd_kernel<<<numBlocks, blockSize>>>(d_osd,
			d_src+(height/2-(height/(zoomx<<1)))*width+(width/2-(width/(zoomx<<1))),
			d_dst, d_dst+planeSize, d_dst+planeSize+(planeSize>>2), width, height,zoomx>>1,
			yColor, uColor, vColor);
	hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachHost);
	hipStreamSynchronize(NULL);

	if (!srcIsMapped) {
		hipFree(d_src);
	}
	if (!osdIsMapped) {
		hipFree(d_osd);
	}
	if (!dstIsMapped) {
		hipMemcpy(dst, d_dst, planeSize * 3/2, hipMemcpyDeviceToHost);
		hipFree(d_dst);
	}
}

//////////////////////////////////////////////////////////////////////
//        YUYVtoI420AndZoomxAndOsd
//
__device__ inline int meanValuePix2_yuyv(unsigned char *img)
{
	return (((int)img[0]+img[2])>>1);
}
__device__ inline int meanValuePix2_yuyv(unsigned char *img, int width)
{
	return (((int)img[0]+img[width<<1])>>1);
}
__device__ inline int meanValuePix4_yuyv(unsigned char *img, int width)
{
	return (((int)img[0]+img[2]+img[width<<1]+img[width<<1+2])>>2);
}
/*
__global__ void gpuConvertYUYVtoI420AndZoomxAndOsd_kernel(unsigned char *osd,unsigned char *src,
		unsigned char *dY,unsigned char *dU,unsigned char *dV,
		unsigned int width, unsigned int height, int zoomxStep,
		int yColor, int uColor, int vColor)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	int zoomIx = idx>>zoomxStep;
	int halfH = height>>1;
	for (int i = 0; i < halfH; ++i) {
		int zoomIy = i>>zoomxStep;
		int y00 = meanValuePix4_yuyv(src+zoomIy*2*width*2+zoomIx*4+0, width);
		int cb0 = src[zoomIy*2*width*2+zoomIx*4+1];
		int y01 = meanValuePix4_yuyv(src+zoomIy*2*width*2+zoomIx*4+2, width);
		int cr0 = src[zoomIy*2*width*2+zoomIx*4+3];
		int y10 = meanValuePix4_yuyv(src+(zoomIy*2+1)*width*2+zoomIx*4+0, width);
		int y11 = meanValuePix4_yuyv(src+(zoomIy*2+1)*width*2+zoomIx*4+2, width);

		int a00 = (osd[i*2*width+idx*2+0]+1)>>8;
		int a01 = (osd[i*2*width+idx*2+1]+1)>>8;
		int a10 = (osd[(i*2+1)*width+idx*2+0]+1)>>8;
		int a11 = (osd[(i*2+1)*width+idx*2+1]+1)>>8;
		int auv = (a00 | a01 | a10 | a11);

		dY[i*2*width+idx*2+0] = (1-a00)*y00 + a00*yColor;
		dY[i*2*width+idx*2+1] = (1-a01)*y01 + a01*yColor;
		dY[(i*2+1)*width+idx*2+0] = (1-a10)*y10 + a10*yColor;
		dY[(i*2+1)*width+idx*2+1] = (1-a11)*y11 + a11*yColor;
		dU[i*(width>>1)+idx] = (1-auv)*cb0 + auv*uColor;
		dV[i*(width>>1)+idx] = (1-auv)*cr0 + auv*vColor;
	}
}

__global__ void gpuConvertYUYVtoI420AndZoomxAndOsd_async_kernel(unsigned char *osd,unsigned char *src,
		unsigned char *dY,unsigned char *dU,unsigned char *dV,
		unsigned int width, unsigned int height, unsigned int heightBegin, int zoomxStep,
		int yColor, int uColor, int vColor)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	int zoomIx = idx>>zoomxStep;
	int halfH = height>>1;
	int halfB = (heightBegin>>1);
	for (int i = 0; i < halfH; ++i) {
		int zoomIy = (i+halfB)>>zoomxStep;
		int y00 = meanValuePix4_yuyv(src+zoomIy*2*width*2+zoomIx*4+0, width);
		int cb0 = src[zoomIy*2*width*2+zoomIx*4+1];
		int y01 = meanValuePix4_yuyv(src+zoomIy*2*width*2+zoomIx*4+2, width);
		int cr0 = src[zoomIy*2*width*2+zoomIx*4+3];
		int y10 = meanValuePix4_yuyv(src+(zoomIy*2+1)*width*2+zoomIx*4+0, width);
		int y11 = meanValuePix4_yuyv(src+(zoomIy*2+1)*width*2+zoomIx*4+2, width);

		int a00 = (osd[i*2*width+idx*2+0]+1)>>8;
		int a01 = (osd[i*2*width+idx*2+1]+1)>>8;
		int a10 = (osd[(i*2+1)*width+idx*2+0]+1)>>8;
		int a11 = (osd[(i*2+1)*width+idx*2+1]+1)>>8;
		int auv = (a00 | a01 | a10 | a11);

		dY[i*2*width+idx*2+0] = (1-a00)*y00 + a00*yColor;
		dY[i*2*width+idx*2+1] = (1-a01)*y01 + a01*yColor;
		dY[(i*2+1)*width+idx*2+0] = (1-a10)*y10 + a10*yColor;
		dY[(i*2+1)*width+idx*2+1] = (1-a11)*y11 + a11*yColor;
		dU[i*(width>>1)+idx] = (1-auv)*cb0 + auv*uColor;
		dV[i*(width>>1)+idx] = (1-auv)*cr0 + auv*vColor;
	}
}
*/
__global__ void gpuConvertYUYVtoI420AndZoomx_kernel(unsigned char *src,
		unsigned char *dY,unsigned char *dU,unsigned char *dV,
		unsigned int width, unsigned int height, int zoomxStep)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	int zoomIx = idx>>zoomxStep;
	int halfH = height>>1;
	for (int i = 0; i < halfH; ++i) {
		int zoomIy = i>>zoomxStep;
		int y00 = src[zoomIy*2*width*2+zoomIx*4+0];
		int cb0 = src[zoomIy*2*width*2+zoomIx*4+1];
		int y01 = meanValuePix2_yuyv(src+zoomIy*2*width*2+zoomIx*4+0);
		int cr0 = src[zoomIy*2*width*2+zoomIx*4+3];
		int y10 = meanValuePix2_yuyv(src+zoomIy*2*width*2+zoomIx*4+0, width);
		int y11 = meanValuePix4_yuyv(src+zoomIy*2*width*2+zoomIx*4+0, width);

		dY[i*2*width+idx*2+0] = y00;
		dY[i*2*width+idx*2+1] = y01;
		dY[(i*2+1)*width+idx*2+0] = y10;
		dY[(i*2+1)*width+idx*2+1] = y11;
		dU[i*(width>>1)+idx] = cb0;
		dV[i*(width>>1)+idx] = cr0;
	}
}

__global__ void gpuConvertYUYVtoI420AndZoomxAndOsd_kernel(unsigned char *osd,unsigned char *src,
		unsigned char *dY,unsigned char *dU,unsigned char *dV,
		unsigned int width, unsigned int height, int zoomxStep,
		int yColor, int uColor, int vColor)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	int zoomIx = idx>>zoomxStep;
	int halfH = height>>1;
	for (int i = 0; i < halfH; ++i) {
		int zoomIy = i>>zoomxStep;
		int y00 = src[zoomIy*2*width*2+zoomIx*4+0];
		int cb0 = src[zoomIy*2*width*2+zoomIx*4+1];
		int y01 = meanValuePix2_yuyv(src+zoomIy*2*width*2+zoomIx*4+0);
		int cr0 = src[zoomIy*2*width*2+zoomIx*4+3];
		int y10 = meanValuePix2_yuyv(src+zoomIy*2*width*2+zoomIx*4+0, width);
		int y11 = meanValuePix4_yuyv(src+zoomIy*2*width*2+zoomIx*4+0, width);

		int a00 = (osd[i*2*width+idx*2+0]+1)>>8;
		int a01 = (osd[i*2*width+idx*2+1]+1)>>8;
		int a10 = (osd[(i*2+1)*width+idx*2+0]+1)>>8;
		int a11 = (osd[(i*2+1)*width+idx*2+1]+1)>>8;
		int auv = (a00 | a01 | a10 | a11);

		dY[i*2*width+idx*2+0] = (1-a00)*y00 + a00*yColor;
		dY[i*2*width+idx*2+1] = (1-a01)*y01 + a01*yColor;
		dY[(i*2+1)*width+idx*2+0] = (1-a10)*y10 + a10*yColor;
		dY[(i*2+1)*width+idx*2+1] = (1-a11)*y11 + a11*yColor;
		dU[i*(width>>1)+idx] = (1-auv)*cb0 + auv*uColor;
		dV[i*(width>>1)+idx] = (1-auv)*cr0 + auv*vColor;
	}
}

__global__ void gpuConvertYUYVtoI420AndZoomx_async_kernel(unsigned char *src,
		unsigned char *dY,unsigned char *dU,unsigned char *dV,
		unsigned int width, unsigned int height, unsigned int heightBegin, int zoomxStep)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	int zoomIx = idx>>zoomxStep;
	int halfH = height>>1;
	int halfB = (heightBegin>>1);
	for (int i = 0; i < halfH; ++i) {
		int zoomIy = (i+halfB)>>zoomxStep;
		int y00 = src[zoomIy*2*width*2+zoomIx*4+0];
		int cb0 = src[zoomIy*2*width*2+zoomIx*4+1];
		int y01 = meanValuePix2_yuyv(src+zoomIy*2*width*2+zoomIx*4+0);
		int cr0 = src[zoomIy*2*width*2+zoomIx*4+3];
		int y10 = meanValuePix2_yuyv(src+zoomIy*2*width*2+zoomIx*4+0, width);
		int y11 = meanValuePix4_yuyv(src+zoomIy*2*width*2+zoomIx*4+0, width);

		dY[i*2*width+idx*2+0] = y00;
		dY[i*2*width+idx*2+1] = y01;
		dY[(i*2+1)*width+idx*2+0] = y10;
		dY[(i*2+1)*width+idx*2+1] = y11;
		dU[i*(width>>1)+idx] = cb0;
		dV[i*(width>>1)+idx] = cr0;
	}
}
__global__ void gpuConvertYUYVtoI420AndZoomxAndOsd_async_kernel(unsigned char *osd,unsigned char *src,
		unsigned char *dY,unsigned char *dU,unsigned char *dV,
		unsigned int width, unsigned int height, unsigned int heightBegin, int zoomxStep,
		int yColor, int uColor, int vColor)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	const int zoomIx = idx>>zoomxStep;
	const int halfH = height>>1;
	const int halfB = (heightBegin>>1);
	for (int i = 0; i < halfH; ++i) {
		int zoomIy = (i+halfB)>>zoomxStep;

		int y00 = src[zoomIy*2*width*2+zoomIx*4+0];
		int cb0 = src[zoomIy*2*width*2+zoomIx*4+1];
		int y01 = meanValuePix2_yuyv(src+zoomIy*2*width*2+zoomIx*4+0);
		int cr0 = src[zoomIy*2*width*2+zoomIx*4+3];
		int y10 = meanValuePix2_yuyv(src+zoomIy*2*width*2+zoomIx*4+0, width);
		int y11 = meanValuePix4_yuyv(src+zoomIy*2*width*2+zoomIx*4+0, width);

		int a00 = (osd[i*2*width+idx*2+0]+1)>>8;
		int a01 = (osd[i*2*width+idx*2+1]+1)>>8;
		int a10 = (osd[(i*2+1)*width+idx*2+0]+1)>>8;
		int a11 = (osd[(i*2+1)*width+idx*2+1]+1)>>8;
		int auv = (a00 | a01 | a10 | a11);

		dY[i*2*width+idx*2+0] = (1-a00)*y00 + a00*yColor;
		dY[i*2*width+idx*2+1] = (1-a01)*y01 + a01*yColor;
		dY[(i*2+1)*width+idx*2+0] = (1-a10)*y10 + a10*yColor;
		dY[(i*2+1)*width+idx*2+1] = (1-a11)*y11 + a11*yColor;
		dU[i*(width>>1)+idx] = (1-auv)*cb0 + auv*uColor;
		dV[i*(width>>1)+idx] = (1-auv)*cr0 + auv*vColor;
	}
}
void gpuConvertYUYVtoI420AndZoomxAndOsd(unsigned char  *osd,unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height, int zoomx, int colorYUV)
{
	unsigned char *d_src = NULL;
	unsigned char *d_dst = NULL;
	unsigned char *d_osd = NULL;
	size_t planeSize = width * height * sizeof(unsigned char);

	unsigned int flags;
	bool srcIsMapped = (hipHostGetFlags(&flags, src) == hipSuccess) && (flags & hipHostMallocMapped);
	bool dstIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) && (flags & hipHostMallocMapped);
	bool osdIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) && (flags & hipHostMallocMapped);

	if (srcIsMapped) {
		d_src = src;
		hipStreamAttachMemAsync(NULL, src, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_src, planeSize * 2);
		hipMemcpy(d_src, src, planeSize * 2, hipMemcpyHostToDevice);
	}
	if (osdIsMapped) {
		d_osd = osd;
		hipStreamAttachMemAsync(NULL, osd, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_osd, planeSize * 1);
		hipMemcpy(d_osd, osd, planeSize * 1, hipMemcpyHostToDevice);
	}
	if (dstIsMapped) {
		d_dst = dst;
		hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_dst, planeSize * 3/2);
	}

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	int yColor = GETYVAL(colorYUV);
	int uColor = GETUVAL(colorYUV);
	int vColor = GETVVAL(colorYUV);
	gpuConvertYUYVtoI420AndZoomxAndOsd_kernel<<<numBlocks, blockSize>>>(d_osd,
			d_src+(height/2-(height/(zoomx<<1)))*width*2+(width/2-(width/(zoomx<<1)))*2,
			d_dst, d_dst+planeSize, d_dst+planeSize+(planeSize>>2), width, height,zoomx>>1,
			yColor, uColor, vColor);
	hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachHost);
	hipStreamSynchronize(NULL);

	if (!srcIsMapped) {
		hipFree(d_src);
	}
	if (!osdIsMapped) {
		hipFree(d_osd);
	}
	if (!dstIsMapped) {
		hipMemcpy(dst, d_dst, planeSize * 3/2, hipMemcpyDeviceToHost);
		hipFree(d_dst);
	}
}

////////////////////////////////////////////////////////////////////
//
//   040_app
//

#define STREAM_CNT	(4)
#define CHANNELS_MAX (5)
#define WIDTH_MAX	(1920)
#define HEIGHT_MAX  (1080)

#define DMEM_CNT	(3)
typedef struct _CUOBJ{
	int index;
	bool bAlloc;
	hipEvent_t start, stop;
	hipStream_t streams[STREAM_CNT];
	unsigned char *d_src;
	unsigned char *d_osd;
	unsigned char *d_mem[DMEM_CNT];
	float elapsedTimeMax, elapsedTimeMin, elapsedTimeSum;
	unsigned int count;
	int imem;
	int64 stampBegin;
	int64 stampEnd;
}CUOBJ, *PCUOBJ;
static CUOBJ gObjs[CHANNELS_MAX];
static OSA_MutexHndl *mutex = NULL;
static bool bCreateMutex = false;
#define LOCK     OSA_mutexLock(mutex);
#define UNLOCK   OSA_mutexUnlock(mutex);
static hipError_t cuConvert_yuyvToI420andOsd(PCUOBJ pObj, Mat src, Mat osd, Mat &dst, int colorYUV);
static hipError_t cuConvert_yuyvToI420AndZoomxAndOsd(PCUOBJ pObj, Mat src, Mat osd, Mat &dst, int zoomx, int colorYUV);
static hipError_t cuConvert_yuyvToI420andOsd_async(PCUOBJ pObj, Mat src, Mat osd, Mat &dst, int colorYUV);
static hipError_t cuConvert_yuyvToI420AndZoomxAndOsd_async(PCUOBJ pObj, Mat src, Mat osd, Mat &dst, int zoomx, int colorYUV);
static hipError_t cuConvert_yuyvToI420AndZoomx_async(PCUOBJ pObj, Mat src, Mat &dst, int zoomx);
static hipError_t cuConvert_grayToI420andOsd(PCUOBJ pObj, Mat src, Mat osd, Mat &dst, int colorYUV);
static hipError_t cuConvert_grayToI420AndZoomxAndOsd(PCUOBJ pObj, Mat src, Mat osd, Mat &dst, int zoomx, int colorYUV);
static hipError_t cuConvert_grayToI420andOsd_async(PCUOBJ pObj, Mat src, Mat osd, Mat &dst, int colorYUV);
static hipError_t cuConvert_grayToI420AndZoomxAndOsd_async(PCUOBJ pObj, Mat src, Mat osd, Mat &dst, int zoomx, int colorYUV);
static hipError_t cuConvert_grayToI420AndZoomx_async(PCUOBJ pObj, Mat src, Mat &dst, int zoomx);

hipError_t cuConvertInit(int nChannels, OSA_MutexHndl *mutexLock)
{
	hipError_t ret = hipSuccess;
	int count, i;
	hipDeviceProp_t prop;
	size_t planeSize = WIDTH_MAX * HEIGHT_MAX;

	if(nChannels<=0 || nChannels >CHANNELS_MAX)
		nChannels = CHANNELS_MAX;

	memset(&gObjs, 0, sizeof(gObjs));

	ret = hipGetDeviceCount(&count);

	ret = hipGetDeviceProperties(&prop, 0);

	printf("   --- General Information for GPU device ---\n");
	printf("Name:                  %s\n", prop.name);
	printf("Capability:            %d.%d\n", prop.major, prop.minor);
	printf("Clock rate:            %d\n", prop.clockRate);
	printf("Copy overlap:          %d\n", prop.deviceOverlap);
	printf("Total global mem:      %ld\n", prop.totalGlobalMem);
	printf("Total constant mem:    %ld\n", prop.totalConstMem);
	printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
	printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
	printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("Max grid dimensions:   (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);

    const NppLibraryVersion *libVer   = nppGetLibVersion();
    printf("NPP Library Version %d.%d.%d\n", libVer->major, libVer->minor, libVer->build);

	int driverVersion, runtimeVersion;
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
	printf("CUDA Driver  Version: %d.%d\n", driverVersion/1000, (driverVersion%100)/10);
	printf("CUDA Runtime Version: %d.%d\n", runtimeVersion/1000, (runtimeVersion%100)/10);

	bool bVal = checkCudaCapabilities(1, 0);

	for(int chId = 0; chId<nChannels; chId++)
	{
		gObjs[chId].index = chId;
		ret = hipEventCreate(&gObjs[chId].start);
		ret = hipEventCreate(&gObjs[chId].stop);

		for(i=0; i<STREAM_CNT; i++)
			ret = hipStreamCreate(&gObjs[chId].streams[i]);

		ret = hipMalloc(&gObjs[chId].d_src, planeSize * 3);
		ret = hipMalloc(&gObjs[chId].d_osd, planeSize * 4);
		for(i=0; i<DMEM_CNT; i++)
			ret = hipMalloc(&gObjs[chId].d_mem[i], planeSize * 3);
		gObjs[chId].bAlloc = true;
		gObjs[chId].elapsedTimeMin = 1000.0f;
	}

	if(mutexLock != NULL)
		mutex = mutexLock;
	if(mutex == NULL){
		mutex = new OSA_MutexHndl;
		OSA_mutexCreate(mutex);
		bCreateMutex = true;
	}

	return ret;
}

hipError_t cuConvertUinit()
{
	int i;

	OSA_mutexUnlock(mutex);
	OSA_mutexLock(mutex);

	for(int chId=0; chId<CHANNELS_MAX; chId++)
	{
		if(!gObjs[chId].bAlloc)
			break;
		for(i=0; i<STREAM_CNT; i++){
			hipStreamDestroy(gObjs[chId].streams[i]);
		}
		hipEventDestroy(gObjs[chId].start);
		hipEventDestroy(gObjs[chId].stop);

		hipFree(gObjs[chId].d_src);
		hipFree(gObjs[chId].d_osd);
		for(i=0; i<DMEM_CNT; i++)
			hipFree(gObjs[chId].d_mem[i]);

		gObjs[chId].bAlloc = false;
	}
	if(bCreateMutex){
		OSA_mutexDelete(mutex);
		delete mutex;
		bCreateMutex = false;
	}
	mutex = NULL;

	return hipSuccess;
}

static hipError_t cuConvert_yuyvToI420andOsd(PCUOBJ pObj, Mat src, Mat osd, Mat &dst, int colorYUV)
{
	hipError_t ret = hipSuccess;
	int width = src.cols, height = src.rows;
	size_t planeSize = width * height;
	int yColor = GETYVAL(colorYUV);
	int uColor = GETUVAL(colorYUV);
	int vColor = GETVVAL(colorYUV);

	pObj->imem = 0;
	unsigned char *d_src = pObj->d_src;
	unsigned char *d_dst = pObj->d_mem[pObj->imem];
	unsigned char *d_osd = pObj->d_osd;

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;

	ret = hipMemcpy(d_src, src.data, planeSize*2, hipMemcpyHostToDevice);
	ret = hipMemcpy(d_osd, osd.data, planeSize, hipMemcpyHostToDevice);
	gpuConvertYUYVtoI420AndOsd_kernel<<<numBlocks, blockSize>>>(
			d_osd, d_src, d_dst,
			d_dst+planeSize,
			d_dst+planeSize+(planeSize>>2),
			width, height,yColor, uColor, vColor);
	hipStreamSynchronize(NULL);
	ret = hipMemcpy(dst.data,d_dst, planeSize*3/2, hipMemcpyDeviceToHost);

	return ret;
}

static hipError_t cuConvert_grayToI420andOsd(PCUOBJ pObj, Mat src, Mat osd, Mat &dst, int colorYUV)
{
	hipError_t ret = hipSuccess;
	int width = src.cols, height = src.rows;
	size_t planeSize = width * height;
	int yColor = GETYVAL(colorYUV);
	int uColor = GETUVAL(colorYUV);
	int vColor = GETVVAL(colorYUV);

	pObj->imem = 0;
	unsigned char *d_src = pObj->d_src;
	unsigned char *d_dst = pObj->d_mem[pObj->imem];
	unsigned char *d_osd = pObj->d_osd;

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;

	ret = hipMemcpy(d_src, src.data, planeSize, hipMemcpyHostToDevice);
	ret = hipMemcpy(d_osd, osd.data, planeSize, hipMemcpyHostToDevice);
	gpuConvertGRAYtoI420AndOsd_kernel<<<numBlocks, blockSize>>>(
			d_osd, d_src, d_dst,
			d_dst+planeSize,
			d_dst+planeSize+(planeSize>>2),
			width, height, yColor, uColor, vColor);
	hipStreamSynchronize(NULL);
	ret = hipMemcpy(dst.data,d_dst, planeSize*3/2, hipMemcpyDeviceToHost);

	return ret;
}

static hipError_t cuConvert_yuyvToI420AndZoomxAndOsd(PCUOBJ pObj, Mat src, Mat osd, Mat &dst, int zoomx, int colorYUV)
{
	hipError_t ret = hipSuccess;
	int width = src.cols, height = src.rows;
	size_t planeSize = width * height;
	int yColor = GETYVAL(colorYUV);
	int uColor = GETUVAL(colorYUV);
	int vColor = GETVVAL(colorYUV);

	pObj->imem = 0;
	unsigned char *d_src = pObj->d_src;
	unsigned char *d_dst = pObj->d_mem[pObj->imem];
	unsigned char *d_osd = pObj->d_osd;

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;

	ret = hipMemcpy(d_src, src.data, planeSize*2, hipMemcpyHostToDevice);
	ret = hipMemcpy(d_osd, osd.data, planeSize, hipMemcpyHostToDevice);
	if(zoomx<=1){
		gpuConvertYUYVtoI420AndOsd_kernel<<<numBlocks, blockSize>>>(
				d_osd, d_src, d_dst, d_dst+planeSize, d_dst+planeSize+(planeSize>>2),
				width, height,yColor, uColor, vColor);
	}else{
		gpuConvertYUYVtoI420AndZoomxAndOsd_kernel<<<numBlocks, blockSize>>>(
				d_osd, d_src+(int)((float)height/2-((float)height/(zoomx<<1))+0.5f)*width*2+(width/2-(width/(zoomx<<1)))*2,
				d_dst, d_dst+planeSize, d_dst+planeSize+(planeSize>>2),
				width, height, zoomx>>1, yColor, uColor, vColor);
	}
	hipStreamSynchronize(NULL);
	ret = hipMemcpy(dst.data,d_dst, planeSize*3/2, hipMemcpyDeviceToHost);

	return ret;
}

static hipError_t cuConvert_grayToI420AndZoomxAndOsd(PCUOBJ pObj, Mat src, Mat osd, Mat &dst, int zoomx, int colorYUV)
{
	hipError_t ret = hipSuccess;
	int width = src.cols, height = src.rows;
	size_t planeSize = width * height;
	int yColor = GETYVAL(colorYUV);
	int uColor = GETUVAL(colorYUV);
	int vColor = GETVVAL(colorYUV);

	pObj->imem = 0;
	unsigned char *d_src = pObj->d_src;
	unsigned char *d_dst = pObj->d_mem[pObj->imem];
	unsigned char *d_osd = pObj->d_osd;

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;

	ret = hipMemcpy(d_src, src.data, planeSize, hipMemcpyHostToDevice);
	ret = hipMemcpy(d_osd, osd.data, planeSize, hipMemcpyHostToDevice);
	if(zoomx<=1){
		gpuConvertGRAYtoI420AndOsd_kernel<<<numBlocks, blockSize>>>(
				d_osd, d_src, d_dst,
				d_dst+planeSize,
				d_dst+planeSize+(planeSize>>2),
				width, height, yColor, uColor, vColor);
	}else{
		gpuConvertGRAYtoI420AndZoomxAndOsd_kernel<<<numBlocks, blockSize>>>(
				d_osd, d_src+(height/2-(height/(zoomx<<1)))*width+(width/2-(width/(zoomx<<1))),
				d_dst, d_dst+planeSize, d_dst+planeSize+(planeSize>>2),
				width, height, zoomx>>1, yColor, uColor, vColor);
	}
	hipStreamSynchronize(NULL);
	ret = hipMemcpy(dst.data,d_dst, planeSize*3/2, hipMemcpyDeviceToHost);

	return ret;
}

static hipError_t cuConvert_yuyvToI420AndZoomxAndOsd_async(PCUOBJ pObj, Mat src, Mat osd, Mat &dst, int zoomx, int colorYUV)
{
	hipError_t ret = hipSuccess;
	int i, width = src.cols, height = src.rows;
	size_t planeSize = width * height;
	int yColor = GETYVAL(colorYUV);
	int uColor = GETUVAL(colorYUV);
	int vColor = GETVVAL(colorYUV);

	pObj->imem = 0;
	unsigned char *d_src = pObj->d_src;
	unsigned char *d_dst = pObj->d_mem[pObj->imem];
	unsigned char *d_osd = pObj->d_osd;

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	unsigned int nBCs = (src.rows * src.cols * src.channels())/STREAM_CNT;
	unsigned int nBCo = (osd.rows * osd.cols * osd.channels())/STREAM_CNT;
	unsigned int nBCd = (src.rows * src.cols)/STREAM_CNT;//I420
	unsigned int nBCduv = (src.rows/2) * ((src.cols/2)/STREAM_CNT);

	if(zoomx<=1){
		for(i = 0; i<STREAM_CNT; i++){
			ret = hipMemcpyAsync(d_src + nBCs*i,
					src.data + nBCs*i, nBCs, hipMemcpyHostToDevice, pObj->streams[i]);
		}
		for(i = 0; i<STREAM_CNT; i++){
			ret = hipMemcpyAsync(d_osd + nBCo*i,
					osd.data + nBCo*i, nBCo, hipMemcpyHostToDevice, pObj->streams[i]);
		}
		for(i = 0; i<STREAM_CNT; i++){
			gpuConvertYUYVtoI420AndOsd_kernel<<<numBlocks, blockSize, 0, pObj->streams[i]>>>(
					d_osd + nBCo*i, d_src + nBCs*i, d_dst + nBCd*i,
					d_dst+planeSize+nBCduv*i,
					d_dst+planeSize+(planeSize>>2)+nBCduv*i,
					width, height/STREAM_CNT,yColor, uColor, vColor);
		}
	}else{
		ret = hipMemcpyAsync(d_src + planeSize*src.channels()/4,
				src.data + planeSize*src.channels()/4, planeSize*src.channels()/4,
				hipMemcpyHostToDevice, pObj->streams[0]);
		ret = hipMemcpyAsync(d_src + planeSize*src.channels()/2,
				src.data + planeSize*src.channels()/2, planeSize*src.channels()/4,
				hipMemcpyHostToDevice, pObj->streams[1]);
		for(i = 0; i<STREAM_CNT; i++){
			ret = hipMemcpyAsync(d_osd + nBCo*i,
					osd.data + nBCo*i, nBCo, hipMemcpyHostToDevice, pObj->streams[i]);
		}
		for(i = 0; i<STREAM_CNT; i++){
			//gpuConvertYUYVtoI420AndZoomxAndOsd_async_kernel
			gpuConvertYUYVtoI420AndZoomxAndOsd_async_kernel<<<numBlocks, blockSize, 0, pObj->streams[i]>>>(
					d_osd + nBCo*i,
					d_src+(int)((float)height/2-((float)height/(zoomx<<1))+0.5f)*width*2+(width/2-(width/(zoomx<<1)))*2,
					d_dst + nBCd*i,
					d_dst+planeSize+nBCduv*i,
					d_dst+planeSize+(planeSize>>2)+nBCduv*i,
					width, height/STREAM_CNT, height*i/STREAM_CNT, zoomx>>1,yColor, uColor, vColor);
		}
	}
	for(i = 0; i<STREAM_CNT; i++){
		ret = hipMemcpyAsync(dst.data + nBCd*i,
				d_dst + nBCd*i, nBCd, hipMemcpyDeviceToHost, pObj->streams[i]);
		ret = hipMemcpyAsync(dst.data + planeSize + nBCduv*i,
				d_dst + planeSize + nBCduv*i, nBCduv, hipMemcpyDeviceToHost, pObj->streams[i]);
		ret = hipMemcpyAsync(dst.data + planeSize + (planeSize>>2) + nBCduv*i,
				d_dst + planeSize + (planeSize>>2) + nBCduv*i, nBCduv, hipMemcpyDeviceToHost, pObj->streams[i]);
	}
	for(i = 0; i<STREAM_CNT; i++){
		ret = hipStreamSynchronize(pObj->streams[i]);
	}

	return ret;
}

static hipError_t cuConvert_yuyvToI420andOsd_async(PCUOBJ pObj, Mat src, Mat osd, Mat &dst, int colorYUV)
{
	hipError_t ret = hipSuccess;
	int i, width = src.cols, height = src.rows;
	size_t planeSize = width * height;
	int yColor = GETYVAL(colorYUV);
	int uColor = GETUVAL(colorYUV);
	int vColor = GETVVAL(colorYUV);

	pObj->imem = 0;
	unsigned char *d_src = pObj->d_src;
	unsigned char *d_dst = pObj->d_mem[pObj->imem];
	unsigned char *d_osd = pObj->d_osd;

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	unsigned int nBCs = (src.rows * src.cols * src.channels())/STREAM_CNT;
	unsigned int nBCo = (osd.rows * osd.cols * osd.channels())/STREAM_CNT;
	unsigned int nBCd = (src.rows * src.cols)/STREAM_CNT;//I420
	unsigned int nBCduv = (src.rows/2) * ((src.cols/2)/STREAM_CNT);

	for(i = 0; i<STREAM_CNT; i++){
		ret = hipMemcpyAsync(d_src + nBCs*i, 
				src.data + nBCs*i, nBCs, hipMemcpyHostToDevice, pObj->streams[i]);
	}
	for(i = 0; i<STREAM_CNT; i++){
		ret = hipMemcpyAsync(d_osd + nBCo*i, 
				osd.data + nBCo*i, nBCo, hipMemcpyHostToDevice, pObj->streams[i]);
	}
	for(i = 0; i<STREAM_CNT; i++){
		gpuConvertYUYVtoI420AndOsd_kernel<<<numBlocks, blockSize, 0, pObj->streams[i]>>>(
				d_osd + nBCo*i, d_src + nBCs*i, d_dst + nBCd*i, 
				d_dst+planeSize+nBCduv*i, 
				d_dst+planeSize+(planeSize>>2)+nBCduv*i, 
				width, height/STREAM_CNT,yColor, uColor, vColor);
	}
	for(i = 0; i<STREAM_CNT; i++){
		ret = hipMemcpyAsync(dst.data + nBCd*i,
				d_dst + nBCd*i, nBCd, hipMemcpyDeviceToHost, pObj->streams[i]);
		ret = hipMemcpyAsync(dst.data + planeSize + nBCduv*i,
				d_dst + planeSize + nBCduv*i, nBCduv, hipMemcpyDeviceToHost, pObj->streams[i]);
		ret = hipMemcpyAsync(dst.data + planeSize + (planeSize>>2) + nBCduv*i,
				d_dst + planeSize + (planeSize>>2) + nBCduv*i, nBCduv, hipMemcpyDeviceToHost, pObj->streams[i]);
	}
	for(i = 0; i<STREAM_CNT; i++){
		ret = hipStreamSynchronize(pObj->streams[i]);
	}

	return ret;
}

static hipError_t cuConvert_yuyvToI420AndZoomx_async(PCUOBJ pObj, Mat src, Mat &dst, int zoomx)
{
	hipError_t ret = hipSuccess;
	int i, width = src.cols, height = src.rows;
	size_t planeSize = width * height;

	pObj->imem = 0;
	unsigned char *d_src = pObj->d_src;
	unsigned char *d_dst = pObj->d_mem[pObj->imem];

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	unsigned int nBCs = (src.rows * src.cols * src.channels())/STREAM_CNT;
	unsigned int nBCd = (src.rows * src.cols)/STREAM_CNT;//I420
	unsigned int nBCduv = (src.rows/2) * ((src.cols/2)/STREAM_CNT);

	if(zoomx<=1){
		for(i = 0; i<STREAM_CNT; i++){
			ret = hipMemcpyAsync(d_src + nBCs*i,
					src.data + nBCs*i, nBCs, hipMemcpyHostToDevice, pObj->streams[i]);
		}
		for(i = 0; i<STREAM_CNT; i++){
			gpuConvertYUYVtoI420_kernel<<<numBlocks, blockSize, 0, pObj->streams[i]>>>(
					d_src + nBCs*i, d_dst + nBCd*i,
					d_dst+planeSize+nBCduv*i,
					d_dst+planeSize+(planeSize>>2)+nBCduv*i,
					width, height/STREAM_CNT);
		}
	}else{
		ret = hipMemcpyAsync(d_src + planeSize*src.channels()/4,
				src.data + planeSize*src.channels()/4, planeSize*src.channels()/4,
				hipMemcpyHostToDevice, pObj->streams[0]);
		ret = hipMemcpyAsync(d_src + planeSize*src.channels()/2,
				src.data + planeSize*src.channels()/2, planeSize*src.channels()/4,
				hipMemcpyHostToDevice, pObj->streams[1]);
		for(i = 0; i<STREAM_CNT; i++){
			//gpuConvertYUYVtoI420AndZoomxAndOsd_async_kernel
			gpuConvertYUYVtoI420AndZoomx_async_kernel<<<numBlocks, blockSize, 0, pObj->streams[i]>>>(
					d_src+(int)((float)height/2-((float)height/(zoomx<<1))+0.5f)*width*2+(width/2-(width/(zoomx<<1)))*2,
					d_dst + nBCd*i,
					d_dst+planeSize+nBCduv*i,
					d_dst+planeSize+(planeSize>>2)+nBCduv*i,
					width, height/STREAM_CNT, height*i/STREAM_CNT, zoomx>>1);
		}
	}
	for(i = 0; i<STREAM_CNT; i++){
		ret = hipMemcpyAsync(dst.data + nBCd*i,
				d_dst + nBCd*i, nBCd, hipMemcpyDeviceToHost, pObj->streams[i]);
		ret = hipMemcpyAsync(dst.data + planeSize + nBCduv*i,
				d_dst + planeSize + nBCduv*i, nBCduv, hipMemcpyDeviceToHost, pObj->streams[i]);
		ret = hipMemcpyAsync(dst.data + planeSize + (planeSize>>2) + nBCduv*i,
				d_dst + planeSize + (planeSize>>2) + nBCduv*i, nBCduv, hipMemcpyDeviceToHost, pObj->streams[i]);
	}
	for(i = 0; i<STREAM_CNT; i++){
		ret = hipStreamSynchronize(pObj->streams[i]);
	}

	return ret;
}

static hipError_t cuConvert_grayToI420andOsd_async(PCUOBJ pObj, Mat src, Mat osd, Mat &dst, int colorYUV)
{
	hipError_t ret = hipSuccess;
	int i, width = src.cols, height = src.rows;
	size_t planeSize = width * height;
	int yColor = GETYVAL(colorYUV);
	int uColor = GETUVAL(colorYUV);
	int vColor = GETVVAL(colorYUV);

	pObj->imem = 0;
	unsigned char *d_src = pObj->d_src;
	unsigned char *d_dst = pObj->d_mem[pObj->imem];
	unsigned char *d_osd = pObj->d_osd;

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	unsigned int nBCs = (src.rows * src.cols * src.channels())/STREAM_CNT;
	unsigned int nBCo = (osd.rows * osd.cols * osd.channels())/STREAM_CNT;
	unsigned int nBCd = (src.rows * src.cols)/STREAM_CNT;//I420
	unsigned int nBCduv = (src.rows/2) * ((src.cols/2)/STREAM_CNT);

	for(i = 0; i<STREAM_CNT; i++){
		ret = hipMemcpyAsync(d_src + nBCs*i, 
				src.data + nBCs*i, nBCs, hipMemcpyHostToDevice, pObj->streams[i]);
	}
	for(i = 0; i<STREAM_CNT; i++){
		ret = hipMemcpyAsync(d_osd + nBCo*i, 
				osd.data + nBCo*i, nBCo, hipMemcpyHostToDevice, pObj->streams[i]);
	}
	for(i = 0; i<STREAM_CNT; i++){
		gpuConvertGRAYtoI420AndOsd_kernel<<<numBlocks, blockSize, 0, pObj->streams[i]>>>(
				d_osd + nBCo*i, d_src + nBCs*i, d_dst + nBCd*i, 
				d_dst+planeSize+nBCduv*i, 
				d_dst+planeSize+(planeSize>>2)+nBCduv*i, 
				width, height/STREAM_CNT, yColor, uColor, vColor);
	}
	for(i = 0; i<STREAM_CNT; i++){
		ret = hipMemcpyAsync(dst.data + nBCd*i,
				d_dst + nBCd*i, nBCd, hipMemcpyDeviceToHost, pObj->streams[i]);
		ret = hipMemcpyAsync(dst.data + planeSize + nBCduv*i,
				d_dst + planeSize + nBCduv*i, nBCduv, hipMemcpyDeviceToHost, pObj->streams[i]);
		ret = hipMemcpyAsync(dst.data + planeSize + (planeSize>>2) + nBCduv*i,
				d_dst + planeSize + (planeSize>>2) + nBCduv*i, nBCduv, hipMemcpyDeviceToHost, pObj->streams[i]);
	}
	for(i = 0; i<STREAM_CNT; i++){
		ret = hipStreamSynchronize(pObj->streams[i]);
	}

	return ret;
}

static hipError_t cuConvert_grayToI420AndZoomxAndOsd_async(PCUOBJ pObj, Mat src, Mat osd, Mat &dst, int zoomx, int colorYUV)
{
	hipError_t ret = hipSuccess;
	int i, width = src.cols, height = src.rows;
	size_t planeSize = width * height;
	int yColor = GETYVAL(colorYUV);
	int uColor = GETUVAL(colorYUV);
	int vColor = GETVVAL(colorYUV);

	pObj->imem = 0;
	unsigned char *d_src = pObj->d_src;
	unsigned char *d_dst = pObj->d_mem[pObj->imem];
	unsigned char *d_osd = pObj->d_osd;

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	unsigned int nBCs = (src.rows * src.cols * src.channels())/STREAM_CNT;
	unsigned int nBCo = (osd.rows * osd.cols * osd.channels())/STREAM_CNT;
	unsigned int nBCd = (src.rows * src.cols)/STREAM_CNT;//I420
	unsigned int nBCduv = (src.rows/2) * ((src.cols/2)/STREAM_CNT);

	if(zoomx<=1)
	{
		for(i = 0; i<STREAM_CNT; i++){
			ret = hipMemcpyAsync(d_src + nBCs*i,
					src.data + nBCs*i, nBCs, hipMemcpyHostToDevice, pObj->streams[i]);
		}
		for(i = 0; i<STREAM_CNT; i++){
			ret = hipMemcpyAsync(d_osd + nBCo*i,
					osd.data + nBCo*i, nBCo, hipMemcpyHostToDevice, pObj->streams[i]);
		}
		for(i = 0; i<STREAM_CNT; i++){
			gpuConvertGRAYtoI420AndOsd_kernel<<<numBlocks, blockSize, 0, pObj->streams[i]>>>(
					d_osd + nBCo*i, d_src + nBCs*i, d_dst + nBCd*i,
					d_dst+planeSize+nBCduv*i,
					d_dst+planeSize+(planeSize>>2)+nBCduv*i,
					width, height/STREAM_CNT, yColor, uColor, vColor);
		}
	}
	else
	{
		ret = hipMemcpyAsync(d_src + planeSize*src.channels()/4,
				src.data + planeSize*src.channels()/4, planeSize*src.channels()/4,
				hipMemcpyHostToDevice, pObj->streams[0]);
		ret = hipMemcpyAsync(d_src + planeSize*src.channels()/2,
				src.data + planeSize*src.channels()/2, planeSize*src.channels()/4,
				hipMemcpyHostToDevice, pObj->streams[1]);
		for(i = 0; i<STREAM_CNT; i++){
			ret = hipMemcpyAsync(d_osd + nBCo*i,
					osd.data + nBCo*i, nBCo, hipMemcpyHostToDevice, pObj->streams[i]);
		}
		for(i = 0; i<STREAM_CNT; i++){
			gpuConvertGRAYtoI420AndZoomxAndOsd_async_kernel<<<numBlocks, blockSize, 0, pObj->streams[i]>>>(
					d_osd + nBCo*i,
					d_src+(height/2-(height/(zoomx<<1)))*width+(width/2-(width/(zoomx<<1))),
					d_dst+nBCd*i,
					d_dst+planeSize+nBCduv*i,
					d_dst+planeSize+(planeSize>>2)+nBCduv*i,
					width, height/STREAM_CNT, height*i/STREAM_CNT, zoomx>>1,yColor, uColor, vColor);
		}
	}
	for(i = 0; i<STREAM_CNT; i++){
		ret = hipMemcpyAsync(dst.data + nBCd*i,
				d_dst + nBCd*i, nBCd, hipMemcpyDeviceToHost, pObj->streams[i]);
		ret = hipMemcpyAsync(dst.data + planeSize + nBCduv*i,
				d_dst + planeSize + nBCduv*i, nBCduv, hipMemcpyDeviceToHost, pObj->streams[i]);
		ret = hipMemcpyAsync(dst.data + planeSize + (planeSize>>2) + nBCduv*i,
				d_dst + planeSize + (planeSize>>2) + nBCduv*i, nBCduv, hipMemcpyDeviceToHost, pObj->streams[i]);
	}
	for(i = 0; i<STREAM_CNT; i++){
		ret = hipStreamSynchronize(pObj->streams[i]);
	}

	return ret;
}

static hipError_t cuConvert_grayToI420AndZoomx_async(PCUOBJ pObj, Mat src, Mat &dst, int zoomx)
{
	hipError_t ret = hipSuccess;
	int i, width = src.cols, height = src.rows;
	size_t planeSize = width * height;

	pObj->imem = 0;
	unsigned char *d_src = pObj->d_src;
	unsigned char *d_dst = pObj->d_mem[pObj->imem];

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	unsigned int nBCs = (src.rows * src.cols * src.channels())/STREAM_CNT;
	unsigned int nBCd = (src.rows * src.cols)/STREAM_CNT;//I420
	unsigned int nBCduv = (src.rows/2) * ((src.cols/2)/STREAM_CNT);

	if(zoomx<=1)
	{
		for(i = 0; i<STREAM_CNT; i++){
			ret = hipMemcpyAsync(d_src + nBCs*i,
					src.data + nBCs*i, nBCs, hipMemcpyHostToDevice, pObj->streams[i]);
		}
		for(i = 0; i<STREAM_CNT; i++){
			gpuConvertGRAYtoI420_kernel<<<numBlocks, blockSize, 0, pObj->streams[i]>>>(
					d_src + nBCs*i, d_dst + nBCd*i,
					d_dst+planeSize+nBCduv*i,
					d_dst+planeSize+(planeSize>>2)+nBCduv*i,
					width, height/STREAM_CNT);
		}
	}
	else
	{
		ret = hipMemcpyAsync(d_src + planeSize*src.channels()/4,
				src.data + planeSize*src.channels()/4, planeSize*src.channels()/4,
				hipMemcpyHostToDevice, pObj->streams[0]);
		ret = hipMemcpyAsync(d_src + planeSize*src.channels()/2,
				src.data + planeSize*src.channels()/2, planeSize*src.channels()/4,
				hipMemcpyHostToDevice, pObj->streams[1]);
		for(i = 0; i<STREAM_CNT; i++){
			gpuConvertGRAYtoI420AndZoomx_async_kernel<<<numBlocks, blockSize, 0, pObj->streams[i]>>>(
					d_src+(height/2-(height/(zoomx<<1)))*width+(width/2-(width/(zoomx<<1))),
					d_dst+nBCd*i,
					d_dst+planeSize+nBCduv*i,
					d_dst+planeSize+(planeSize>>2)+nBCduv*i,
					width, height/STREAM_CNT, height*i/STREAM_CNT, zoomx>>1);
		}
	}
	for(i = 0; i<STREAM_CNT; i++){
		ret = hipMemcpyAsync(dst.data + nBCd*i,
				d_dst + nBCd*i, nBCd, hipMemcpyDeviceToHost, pObj->streams[i]);
		ret = hipMemcpyAsync(dst.data + planeSize + nBCduv*i,
				d_dst + planeSize + nBCduv*i, nBCduv, hipMemcpyDeviceToHost, pObj->streams[i]);
		ret = hipMemcpyAsync(dst.data + planeSize + (planeSize>>2) + nBCduv*i,
				d_dst + planeSize + (planeSize>>2) + nBCduv*i, nBCduv, hipMemcpyDeviceToHost, pObj->streams[i]);
	}
	for(i = 0; i<STREAM_CNT; i++){
		ret = hipStreamSynchronize(pObj->streams[i]);
	}

	return ret;
}

/*
#define DEBUGTM_begin(c)	float elapsedTime;{	\
	if(c==chId){ret = hipEventRecord(gObjs[chId].start, gObjs[chId].streams[0]);}	\
}
#define DEBUGTM_end(c){	\
	if(c==chId){	\
		ret = hipEventRecord(gObjs[chId].stop, gObjs[chId].streams[0]);	\
		ret = hipEventElapsedTime(&elapsedTime, gObjs[chId].start, gObjs[chId].stop);	\
		gObjs[chId].count ++;	\
		gObjs[chId].elapsedTimeMax = max(gObjs[chId].elapsedTimeMax, elapsedTime);	\
		gObjs[chId].elapsedTimeMin = min(gObjs[chId].elapsedTimeMin, elapsedTime);	\
		gObjs[chId].elapsedTimeSum += elapsedTime;	\
		if(gObjs[chId].count == 1000){	\
			printf("\n%s: ch%d Time taken %3.1f(%3.1f, %3.1f, %3.1f) ms", __func__, chId,	\
					elapsedTime, gObjs[chId].elapsedTimeMax, gObjs[chId].elapsedTimeMin, gObjs[chId].elapsedTimeSum/gObjs[chId].count);	\
			gObjs[chId].elapsedTimeMax = 0.0f; gObjs[chId].elapsedTimeMin = 1000.0f; gObjs[chId].elapsedTimeSum = 0.0f;	\
			gObjs[chId].count = 0;	\
		}	\
	}	\
}
*/

__inline__ void elapsedTimeCals(int chId, const char *func)
{
	float elapsedTime = (gObjs[chId].stampEnd - gObjs[chId].stampBegin)*1000.f/getTickFrequency();
	gObjs[chId].count ++;
	gObjs[chId].elapsedTimeMax = max(gObjs[chId].elapsedTimeMax, elapsedTime);
	gObjs[chId].elapsedTimeMin = min(gObjs[chId].elapsedTimeMin, elapsedTime);
	gObjs[chId].elapsedTimeSum += elapsedTime;
	if(gObjs[chId].count == 30*10){
		printf("\n[%d]%s: ch%d Time taken %3.1f(%3.1f, %3.1f, %3.1f) ms", OSA_getCurTimeInMsec(), func, chId,
				elapsedTime, gObjs[chId].elapsedTimeMax, gObjs[chId].elapsedTimeMin, gObjs[chId].elapsedTimeSum/gObjs[chId].count);
		fflush(stdout);
		gObjs[chId].elapsedTimeMax = 0.0f; gObjs[chId].elapsedTimeMin = 1000.0f; gObjs[chId].elapsedTimeSum = 0.0f;
		gObjs[chId].count = 0;
	}
}

hipError_t cuConvert(int chId, Mat src, Mat osd, Mat &dst, int zoomx, int colorYUV)
{
	LOCK;
	hipError_t ret = hipSuccess;

	gObjs[chId].stampBegin = getTickCount();

	if(src.channels() == 1){
		//cuConvert_grayToI420andOsd(&gObjs[chId], src, osd, dst, colorYUV);
		cuConvert_grayToI420AndZoomxAndOsd(&gObjs[chId], src, osd, dst, zoomx, colorYUV);
	}else{
		//cuConvert_yuyvToI420andOsd(&gObjs[chId], src, osd, dst, colorYUV);
		cuConvert_yuyvToI420AndZoomxAndOsd(&gObjs[chId], src, osd, dst, zoomx, colorYUV);
	}
	if(ret != hipSuccess){
		printf("%s(%i)  : hipGetLastError() CUDA error: %d\n", __FILE__, __LINE__, (int)hipGetLastError());
	}
	ret = hipDeviceSynchronize();
	gObjs[chId].stampEnd = getTickCount();
	elapsedTimeCals(chId, __func__);

	UNLOCK;
	return ret;
}
#if 0
hipError_t cuConvertEnh(int chId, cv::Mat src, cv::Mat osd, cv::Mat &dst, int zoomx, int colorYUV)
{
	LOCK;
	hipError_t ret = hipSuccess;
	gObjs[chId].stampBegin = getTickCount();

	if(src.channels() == 1){
		//cuConvert_grayToI420andOsd(&gObjs[chId], src, osd, dst, colorYUV);
		cuConvert_grayToI420AndZoomxAndOsd(&gObjs[chId], src, osd, dst, zoomx, colorYUV);
	}else{
		//cuConvert_yuyvToI420andOsd(&gObjs[chId], src, osd, dst, colorYUV);
		cuConvert_yuyvToI420AndZoomxAndOsd(&gObjs[chId], src, osd, dst, zoomx, colorYUV);
	}

	Mat enhSrc(src.rows, src.cols,CV_8UC1, gObjs[chId].d_mem[gObjs[chId].imem]);
	Mat enhDst(src.rows, src.cols,CV_8UC1, gObjs[chId].d_mem[gObjs[chId].imem+1]);
	cuClahe(enhSrc, enhDst, (src.cols==1920)?8:8, (src.cols==1920)?4:8);
	int planeSize = src.rows*src.cols;
	ret = hipMemcpy(gObjs[chId].d_mem[gObjs[chId].imem+1]+planeSize,
			gObjs[chId].d_mem[gObjs[chId].imem]+planeSize, planeSize>>1, hipMemcpyDeviceToDevice);
	ret = hipMemcpy(dst.data,gObjs[chId].d_mem[gObjs[chId].imem+1], planeSize, hipMemcpyDeviceToHost);
	gObjs[chId].imem++;

	if(ret != hipSuccess){
		printf("%s(%i)  : hipGetLastError() CUDA error: %d\n", __FILE__, __LINE__, (int)hipGetLastError());
	}
	ret = hipDeviceSynchronize();
	gObjs[chId].stampEnd = getTickCount();
	elapsedTimeCals(chId, __func__);

	UNLOCK;
	return ret;
}
#endif
hipError_t cuConvert_async(int chId, Mat src, Mat osd, Mat &dst, int zoomx, int colorYUV)
{
	LOCK;
	hipError_t ret = hipSuccess;
	gObjs[chId].stampBegin = getTickCount();
	if(src.channels() == 1){
		ret = cuConvert_grayToI420AndZoomxAndOsd_async(&gObjs[chId], src, osd, dst, zoomx, colorYUV);
	}else{
		ret = cuConvert_yuyvToI420AndZoomxAndOsd_async(&gObjs[chId], src, osd, dst, zoomx, colorYUV);
	}
	if(ret != hipSuccess){
		printf("%s(%i)  : hipGetLastError() CUDA error: %d\n", __FILE__, __LINE__, (int)hipGetLastError());
	}
	ret = hipDeviceSynchronize();
	gObjs[chId].stampEnd = getTickCount();
	elapsedTimeCals(chId, __func__);

	UNLOCK;
	return ret;
}

hipError_t cuConvert_async(int chId, Mat src, Mat &dst, int zoomx)
{
	LOCK;
	hipError_t ret = hipSuccess;
	gObjs[chId].stampBegin = getTickCount();
	if(src.channels() == 1){
		ret = cuConvert_grayToI420AndZoomx_async(&gObjs[chId], src, dst, zoomx);
	}else{
		ret = cuConvert_yuyvToI420AndZoomx_async(&gObjs[chId], src, dst, zoomx);
	}
	if(ret != hipSuccess){
		printf("%s(%i)  : hipGetLastError() CUDA error: %d\n", __FILE__, __LINE__, (int)hipGetLastError());
	}
	ret = hipDeviceSynchronize();
	gObjs[chId].stampEnd = getTickCount();
	elapsedTimeCals(chId, __func__);

	UNLOCK;
	return ret;
}
#if 0
hipError_t cuConvertEnh_async(int chId, Mat src, Mat osd, Mat &dst, int zoomx, int colorYUV)
{
	LOCK;
	hipError_t ret = hipSuccess;
	gObjs[chId].stampBegin = getTickCount();

	if(src.channels() == 1){
		ret = cuConvert_grayToI420AndZoomxAndOsd_async(&gObjs[chId], src, osd, dst, zoomx, colorYUV);
	}else{
		ret = cuConvert_yuyvToI420AndZoomxAndOsd_async(&gObjs[chId], src, osd, dst, zoomx, colorYUV);
	}

	Mat enhSrc(src.rows, src.cols,CV_8UC1, gObjs[chId].d_mem[gObjs[chId].imem]);
	Mat enhDst(src.rows, src.cols,CV_8UC1, gObjs[chId].d_mem[gObjs[chId].imem+1]);
	cuClahe(enhSrc, enhDst, (src.cols==1920)?8:8, (src.cols==1920)?4:8);
	int planeSize = src.rows*src.cols;
	ret = hipMemcpy(gObjs[chId].d_mem[gObjs[chId].imem+1]+planeSize,
			gObjs[chId].d_mem[gObjs[chId].imem]+planeSize, planeSize>>1, hipMemcpyDeviceToDevice);
	ret = hipMemcpy(dst.data,gObjs[chId].d_mem[gObjs[chId].imem+1], planeSize, hipMemcpyDeviceToHost);
	gObjs[chId].imem++;

	if(ret != hipSuccess){
		printf("%s(%i)  : hipGetLastError() CUDA error: %d\n", __FILE__, __LINE__, (int)hipGetLastError());
	}
	ret = hipDeviceSynchronize();
	gObjs[chId].stampEnd = getTickCount();
	elapsedTimeCals(chId, __func__);

	UNLOCK;
	return ret;
}

hipError_t cuConvertEnh_async(int chId, Mat src, Mat &dst, int zoomx)
{
	LOCK;
	hipError_t ret = hipSuccess;
	gObjs[chId].stampBegin = getTickCount();

	if(src.channels() == 1){
		ret = cuConvert_grayToI420AndZoomx_async(&gObjs[chId], src, dst, zoomx);
	}else{
		ret = cuConvert_yuyvToI420AndZoomx_async(&gObjs[chId], src, dst, zoomx);
	}

	Mat enhSrc(src.rows, src.cols,CV_8UC1, gObjs[chId].d_mem[gObjs[chId].imem]);
	Mat enhDst(src.rows, src.cols,CV_8UC1, gObjs[chId].d_mem[gObjs[chId].imem+1]);
	cuClahe(enhSrc, enhDst, (src.cols==1920)?8:8, (src.cols==1920)?4:8);
	int planeSize = src.rows*src.cols;
	ret = hipMemcpy(gObjs[chId].d_mem[gObjs[chId].imem+1]+planeSize,
			gObjs[chId].d_mem[gObjs[chId].imem]+planeSize, planeSize>>1, hipMemcpyDeviceToDevice);
	ret = hipMemcpy(dst.data,gObjs[chId].d_mem[gObjs[chId].imem+1], planeSize, hipMemcpyDeviceToHost);
	gObjs[chId].imem++;

	if(ret != hipSuccess){
		printf("%s(%i)  : hipGetLastError() CUDA error: %d\n", __FILE__, __LINE__, (int)hipGetLastError());
	}
	ret = hipDeviceSynchronize();
	gObjs[chId].stampEnd = getTickCount();
	elapsedTimeCals(chId, __func__);

	UNLOCK;
	return ret;
}

static hipError_t cuConvert_yuv2bgr_yuyv_async(PCUOBJ pObj, Mat src, Mat &dst, int flag)
{
	hipError_t ret = hipSuccess;
	OSA_assert(dst.data != NULL);
	int i, width = src.cols, height = src.rows;

	unsigned char *d_src = pObj->d_src;
	unsigned char *d_dst = (flag == CUT_FLAG_devAlloc) ? dst.data : pObj->d_mem[0];

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	unsigned int nBCs = (src.rows * src.cols * src.channels())/STREAM_CNT;
	unsigned int nBCd = (dst.rows * dst.cols * dst.channels())/STREAM_CNT;

	for(i = 0; i<STREAM_CNT; i++){
		ret = hipMemcpyAsync(d_src + nBCs*i,
				src.data + nBCs*i, nBCs, hipMemcpyHostToDevice, pObj->streams[i]);
	}
	for(i = 0; i<STREAM_CNT; i++){
		gpuConvertYUYVtoBGR_kernel<<<numBlocks, blockSize, 0, pObj->streams[i]>>>(
				d_src + nBCs*i, d_dst + nBCd*i,
				width, height/STREAM_CNT);
	}
	if(dst.data != d_dst)
	{
		for(i = 0; i<STREAM_CNT; i++){
			ret = hipMemcpyAsync(dst.data + nBCd*i,
					d_dst + nBCd*i, nBCd, hipMemcpyDeviceToHost, pObj->streams[i]);
		}
	}
	for(i = 0; i<STREAM_CNT; i++){
		ret = hipStreamSynchronize(pObj->streams[i]);
	}

	return ret;
}

static hipError_t cunppConvert_yuv2bgr_yuyv_async(PCUOBJ pObj, const Mat& src, Mat &dst, int flag)
{
	hipError_t ret = hipSuccess;
	OSA_assert(dst.data != NULL);
	int width = src.cols, height = src.rows;
	unsigned char *d_src = pObj->d_src;
	unsigned char *d_dst = (flag == CUT_FLAG_devAlloc) ? dst.data : pObj->d_mem[0];

	NppiSize oSizeROI;
	oSizeROI.width = width;
	oSizeROI.height = height;
	//npp::ImageCPU_8u_C2 oHostSrc;
	//npp::ImageNPP_8u_C2 oDeviceSrc(oHostSrc);
	hipMemcpy(d_src, src.data, width*2, hipMemcpyHostToDevice);
	NppStatus stat = nppiYCbCr422ToBGR_8u_C2C3R(d_src, width*2, d_dst, width*3, oSizeROI);
	if(stat != NPP_NO_ERROR){
		OSA_printf("%s: stat = %d", __func__, stat);
		OSA_assert(stat == NPP_NO_ERROR);
	}

	if(dst.data != d_dst)
	{
		hipMemcpy(dst.data, d_dst, width*3, hipMemcpyDeviceToHost);
	}

	return ret;
}

hipError_t cuConvert_yuv2bgr_yuyv_async(int chId, Mat src, Mat &dst, int flag)
{
	LOCK;
	hipError_t ret = hipSuccess;
	gObjs[chId].stampBegin = getTickCount();
	ret = cuConvert_yuv2bgr_yuyv_async(&gObjs[chId], src, dst, flag);
	//ret = cunppConvert_yuv2bgr_yuyv_async(&gObjs[chId], src, dst, flag);
	if(ret != hipSuccess){
		printf("%s(%i)  : hipGetLastError() CUDA error: %d\n", __FILE__, __LINE__, (int)hipGetLastError());
	}
	ret = hipDeviceSynchronize();
	gObjs[chId].stampEnd = getTickCount();
	elapsedTimeCals(chId, __func__);
	UNLOCK;
	return ret;
}

hipError_t cuConvertEnh_yuv2bgr_yuyv_async(int chId, Mat src, Mat &dst, int flag)
{
	LOCK;
	hipError_t ret = hipSuccess;
	gObjs[chId].stampBegin = getTickCount();
	Mat enhSrc(src.rows, src.cols,CV_8UC3, gObjs[chId].d_mem[0]);
	ret = cuConvert_yuv2bgr_yuyv_async(&gObjs[chId], src, enhSrc, CUT_FLAG_devAlloc);
	if(ret != hipSuccess){
		printf("%s(%i)  : hipGetLastError() CUDA error: %d\n", __FILE__, __LINE__, (int)hipGetLastError());
	}
	//ret = hipMemcpy(dst.data,enhSrc.data, src.rows*src.cols*dst.channels(),hipMemcpyDeviceToDevice);
	cuClahe(enhSrc, dst, (src.cols==1920)?8:8, (src.cols==1920)?4:8, 3.0f);
	//cuTemporalFilter(dst, dst);
	ret = hipDeviceSynchronize();
	gObjs[chId].stampEnd = getTickCount();
	elapsedTimeCals(chId, __func__);
	UNLOCK;
	return ret;
}

hipError_t cuConvertEnh_gray(int chId, cv::Mat src, cv::Mat &dst, int flag)
{
	LOCK;
	hipError_t ret = hipSuccess;
	gObjs[chId].stampBegin = getTickCount();
	Mat enhSrc(src.rows, src.cols,CV_8UC1, gObjs[chId].d_src);
	ret = hipMemcpy(enhSrc.data, src.data, src.rows*src.cols, hipMemcpyHostToDevice);
	OSA_assert(ret == hipSuccess);
	cuClahe(enhSrc, dst, (src.cols==1920)?8:8, (src.cols==1920)?4:8);
	ret = hipDeviceSynchronize();
	gObjs[chId].stampEnd = getTickCount();
	elapsedTimeCals(chId, __func__);
	UNLOCK;
	return ret;
}

/*****************************************************************
 *
 *  cutColor yuv2bgr i420
 */
__global__ void gpuConvertI420toBGR_kernel(
		unsigned char *src_y, unsigned char *src_u, unsigned char *src_v, unsigned char *dst,
		unsigned int width, unsigned int height)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}
	int height_half = (height>>1);
	int width_half = (width>>1);

	for (int i = 0; i < height_half; ++i) {
		int cb = src_u[i*width_half+idx];
		int cr = src_v[i*width_half+idx];
		int b = DESCALE((cb - 128)*COEFFS_3, 14);
		int g = DESCALE((cb - 128)*COEFFS_2 + (cr - 128)*COEFFS_1, 14);
		int r = DESCALE((cr - 128)*COEFFS_0, 14);
		int y00 = src_y[(i*2+0)*width+idx*2+0];
		int y01 = src_y[(i*2+0)*width+idx*2+1];
		int y10 = src_y[(i*2+1)*width+idx*2+0];
		int y11 = src_y[(i*2+1)*width+idx*2+1];

		dst[(i*2+0)*width*3+idx*6+0] = clip(0, 255, y00 + b);//B
		dst[(i*2+0)*width*3+idx*6+1] = clip(0, 255, y00 + g);//G
		dst[(i*2+0)*width*3+idx*6+2] = clip(0, 255, y00 + r);//R
		dst[(i*2+0)*width*3+idx*6+3] = clip(0, 255, y01 + b);//B
		dst[(i*2+0)*width*3+idx*6+4] = clip(0, 255, y01 + g);//G
		dst[(i*2+0)*width*3+idx*6+5] = clip(0, 255, y01 + r);//R
		dst[(i*2+1)*width*3+idx*6+0] = clip(0, 255, y10 + b);//B
		dst[(i*2+1)*width*3+idx*6+1] = clip(0, 255, y10 + g);//G
		dst[(i*2+1)*width*3+idx*6+2] = clip(0, 255, y10 + r);//R
		dst[(i*2+1)*width*3+idx*6+3] = clip(0, 255, y11 + b);//B
		dst[(i*2+1)*width*3+idx*6+4] = clip(0, 255, y11 + g);//G
		dst[(i*2+1)*width*3+idx*6+5] = clip(0, 255, y11 + r);//R
	}
}
static hipError_t cuConvert_yuv2bgr_i420(PCUOBJ pObj, Mat &dst)
{
	hipError_t ret = hipSuccess;
	int width = dst.cols, height = dst.rows;

	unsigned char *d_src = pObj->d_mem[pObj->imem];
	unsigned char *d_dst = dst.data;

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;

	gpuConvertI420toBGR_kernel<<<numBlocks, blockSize>>>(
			d_src, d_src+width*height, d_src+width*height+((width>>1)*(height>>1)),d_dst,
			width, height);
	hipStreamSynchronize(NULL);

	return ret;
}

hipError_t cuConvertConn_yuv2bgr_i420(int chId, Mat &dst, int flag)
{
	LOCK;
	hipError_t ret = hipSuccess;
	ret = cuConvert_yuv2bgr_i420(&gObjs[chId], dst);
	if(ret != hipSuccess){
		printf("%s(%i)  : hipGetLastError() CUDA error: %d\n", __FILE__, __LINE__, (int)hipGetLastError());
	}
	ret = hipDeviceSynchronize();
	UNLOCK;
	return ret;
}
#endif

